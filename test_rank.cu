#include "rank.cu"
#include <stdint.h>


void test_getLinksStartIndex() {

    uint32_t compressedInLinksCount [6] = { 0, 2, 0, 40, 13, 0 };
    uint64_t compressedInLinksStartIndex [6] = { };
    uint64_t size = getLinksStartIndex(6, compressedInLinksCount, compressedInLinksStartIndex);

    if (size != 55) {
        printf("getLinksStartIndex() wrong composed in links size!\n");
    }

    uint64_t expected [6] = {0,0,2,2,42,55};
    if (std::equal(std::begin(expected), std::end(expected), std::begin(compressedInLinksStartIndex)))
        printf("getLinksStartIndex() works as expected!\n");
    else {
        printf("getLinksStartIndex() doesn't works :(\n");
        for (int i = sizeof(expected) / sizeof(expected[0])-1; i >= 0; i--)
            std::cout << compressedInLinksStartIndex[i] << ' ' << expected[i] << '\n';
    }
}

void test_getCompressedInLinksCount() {

    uint64_t cidsSize = 6;
    uint32_t inLinksCount [6] = { 0, 2, 0, 1, 3, 3 };
    uint64_t inLinksStartIndex [6] = { 0, 0, 2, 2, 3, 6 };
    uint64_t inLinksOuts [] = { 1, 1, 2, 2, 2, 1, 2, 1, 1};
    int outSize = (sizeof(inLinksOuts)/sizeof(*inLinksOuts));

    uint32_t *dev_inLinksCount;
    uint32_t *dev_compressedInLinksCount;
    uint64_t *dev_inLinksStartIndex;
    uint64_t *dev_inLinksOuts;

    hipMalloc(&dev_inLinksCount, cidsSize*sizeof(uint32_t));
    hipMalloc(&dev_compressedInLinksCount, cidsSize*sizeof(uint32_t));
    hipMalloc(&dev_inLinksStartIndex, cidsSize*sizeof(uint64_t));
    hipMalloc(&dev_inLinksOuts, outSize*sizeof(uint64_t));

    hipMemcpy(dev_inLinksCount, inLinksCount, cidsSize*sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_inLinksStartIndex, inLinksStartIndex, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_inLinksOuts, inLinksOuts, outSize*sizeof(uint64_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    getCompressedInLinksCount<<<2,3>>>(
        cidsSize,
        dev_inLinksStartIndex, dev_inLinksCount,
        dev_inLinksOuts, dev_compressedInLinksCount
    );
    hipDeviceSynchronize();

    uint32_t actual[6] = {};
    hipMemcpy(actual, dev_compressedInLinksCount, cidsSize*sizeof(uint32_t), hipMemcpyDeviceToHost);

    uint64_t expected[6] = {0,1,0,1,2,2};
    if (std::equal(std::begin(expected), std::end(expected), std::begin(actual)))
        printf("getCompressedInLinksCount() works as expected!\n");
    else {
       printf("getCompressedInLinksCount() doesn't works :(\n");
       for (int i = sizeof(actual) / sizeof(actual[0])-1; i >= 0; i--)
           std::cout << actual[i] << ' ' << expected[i] << '\n';
    }
}

void test_calculateCidTotalOutStake() {

    int cidsSize = 6;
    int linksSize = 9;
    int usersSize = 3;

    uint32_t outLinksCount [6] = { 0, 2, 0, 1, 3, 3 };
    uint64_t outLinksStartIndex [6] = { 0, 0, 2, 2, 3, 6 };
    uint64_t outLinksUsers [9] = { 1, 0, 2, 0, 2, 1, 2, 1, 0};
    uint64_t stakes [3] = { 1, 2, 3};

    uint32_t *dev_outLinksCount;
    uint64_t *dev_outLinksStartIndex;
    uint64_t *dev_outLinksUsers;
    uint64_t *dev_stakes;
    uint64_t *dev_cidsTotalOutStakes;

    hipMalloc(&dev_outLinksCount, cidsSize*sizeof(uint32_t));
    hipMalloc(&dev_outLinksStartIndex, cidsSize*sizeof(uint64_t));
    hipMalloc(&dev_outLinksUsers, linksSize*sizeof(uint64_t));
    hipMalloc(&dev_stakes, usersSize*sizeof(uint64_t));
    hipMalloc(&dev_cidsTotalOutStakes, cidsSize*sizeof(uint64_t));

    hipMemcpy(dev_outLinksCount, outLinksCount, cidsSize*sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_outLinksStartIndex, outLinksStartIndex, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_outLinksUsers, outLinksUsers, linksSize*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_stakes, stakes, usersSize*sizeof(uint64_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    calculateCidTotalOutStake<<<2,3>>>(
        cidsSize, dev_stakes,
        dev_outLinksStartIndex, dev_outLinksCount,
        dev_outLinksUsers, dev_cidsTotalOutStakes
    );
    hipDeviceSynchronize();

    uint64_t actual[6] = {};
    hipMemcpy(actual, dev_cidsTotalOutStakes, cidsSize*sizeof(uint64_t), hipMemcpyDeviceToHost);

    uint64_t expected[6] = {0,3,0,3,6,6};
    if (std::equal(std::begin(expected), std::end(expected), std::begin(actual)))
        printf("calculateCidTotalOutStake() works as expected!\n");
    else {
       printf("calculateCidTotalOutStake() doesn't works :(\n");
       for (int i = sizeof(actual) / sizeof(actual[0])-1; i >= 0; i--)
           std::cout << actual[i] << ' ' << expected[i] << '\n';
    }
}

void test_find_max_ranks_diff() {

    double prevRank [6] = { -1.324, 32.1, 0.001, 2.231, -3.22, -0.02 };
    double newRank [6] = {1.3242, 32.22, 0.032, 2.231, -3.232, 0.02 };

    double *dev_prevRank;
    double *dev_newRank;
    hipMalloc(&dev_prevRank, 6*sizeof(double));
    hipMalloc(&dev_newRank, 6*sizeof(double));
    hipMemcpy(dev_prevRank, prevRank, 6*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_newRank, newRank, 6*sizeof(double), hipMemcpyHostToDevice);

    double maxDiff = find_max_ranks_diff(dev_prevRank, dev_newRank, 6);
    if (maxDiff == 2.6482)
        printf("find_max_ranks_diff() works as expected!\n");
    else {
       printf("find_max_ranks_diff() doesn't works :(\n");
       std::cout << maxDiff << ' ' << 2.6482 << '\n';
    }
}

void test_getCompressedInLinks() {

    int cidsSize = 8;
    int linksSize = 11;
    int compressedLinksSize = 8;
    int usersSize = 3;

    uint32_t inLinksCount [8] =           {0,0,1,5,4,0,1,0};
    uint32_t compressedInLinksCount [8] = {0,0,1,3,3,0,1,0};
    uint64_t inLinksStartIndex [8] =                {0,0,0,1,6,10,10,11};
    uint64_t compressedInLinksStartIndex [8] =      {0,0,0,1,4,7,7,8};
    uint64_t cidsTotalOutStakes [8] =    {3,3,3,1,6,1,0,3};
    uint64_t inLinksOuts [11]  = {7,1,4,4,4,2,5,0,0,1,3};
    uint64_t inLinksUsers [11] = {0,2,0,1,2,0,1,1,2,1,1};
    uint64_t stakes [3] = {3,1,2};

    uint64_t *dev_inLinksStartIndex;
    uint32_t *dev_inLinksCount;
    uint64_t *dev_cidsTotalOutStakes;
    uint64_t *dev_inLinksOuts;
    uint64_t *dev_inLinksUsers;
    uint64_t *dev_stakes;
    uint64_t *dev_compressedInLinksStartIndex;
    uint32_t *dev_compressedInLinksCount;
    CompressedInLink *dev_compressedInLinks;

    hipMalloc(&dev_inLinksStartIndex, cidsSize*sizeof(uint64_t));
    hipMalloc(&dev_inLinksCount, cidsSize*sizeof(uint32_t));
    hipMalloc(&dev_cidsTotalOutStakes, cidsSize*sizeof(uint64_t));
    hipMalloc(&dev_inLinksOuts, linksSize*sizeof(uint64_t));
    hipMalloc(&dev_inLinksUsers, linksSize*sizeof(uint64_t));
    hipMalloc(&dev_stakes, usersSize*sizeof(uint64_t));
    hipMalloc(&dev_compressedInLinksStartIndex, cidsSize*sizeof(uint64_t));
    hipMalloc(&dev_compressedInLinksCount, cidsSize*sizeof(uint32_t));
    hipMalloc(&dev_compressedInLinks, compressedLinksSize*sizeof(CompressedInLink));

    hipMemcpy(dev_inLinksStartIndex, inLinksStartIndex, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_inLinksCount, inLinksCount, cidsSize*sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_cidsTotalOutStakes, cidsTotalOutStakes, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_inLinksOuts, inLinksOuts, linksSize*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_inLinksUsers, inLinksUsers, linksSize*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_stakes, stakes, usersSize*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_compressedInLinksStartIndex, compressedInLinksStartIndex, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_compressedInLinksCount, compressedInLinksCount, cidsSize*sizeof(uint32_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    getCompressedInLinks<<<4,2>>>(
        cidsSize,
        dev_inLinksStartIndex, dev_inLinksCount, dev_cidsTotalOutStakes,
        dev_inLinksOuts, dev_inLinksUsers,
        dev_stakes,
        dev_compressedInLinksStartIndex, compressedInLinksCount,
        dev_compressedInLinks
    );
    hipDeviceSynchronize();

    CompressedInLink actual[8] = {};
    hipMemcpy(actual, dev_compressedInLinks, compressedLinksSize*sizeof(CompressedInLink), hipMemcpyDeviceToHost);

    CompressedInLink expected[8] = {
        {7,1.0},{1,0.666667},{4,1},{2,1},{5,1},{0,1},{1,0.333333},{3,1}
    };

    printf("calculateCidTotalOutStake() output\n");
    for (int i = sizeof(actual) / sizeof(actual[0])-1; i >= 0; i--) {
       std::cout << actual[i].fromIndex <<'_'<< actual[i].weight << "   ";
       std::cout << expected[i].fromIndex <<'_'<< expected[i].weight << '\n';
    }
}

void test_calculate_rank() {
    uint64_t stakes [4] = {1000000000000000, 1000000000000000, 2000000000000000, 3000000000000000};
    uint64_t stakesSize = 4;
    uint64_t cidsSize = 6;
    uint64_t linksSize = 14;
    uint32_t inLinksCount [6] = {2,2,3,3,2,2};
    uint32_t outLinksCount [6] = {3,2,2,2,3,2};
    uint64_t inLinksOuts [14]  = {4,5,0,4,0,1,4,0,1,2,3,5,2,3};
    uint64_t inLinksUsers [14] = {2,2,2,2,1,3,2,1,1,3,3,1,1,3};
    uint64_t outLinksUsers [14] = {1,2,1,1,3,1,3,3,3,2,2,2,1,2};
    double actualRank[6];

    calculate_rank(
        stakes, stakesSize, cidsSize, linksSize,
        inLinksCount, outLinksCount,
        inLinksOuts, inLinksUsers, outLinksUsers,
        actualRank
    );
    
    double expectedRank [6] = {0.15276519918059792, 0.13283409782427277, 0.1848430041460299, 0.20276579605193035, 0.15308408018111755, 0.14990462558877385};
    
    printf("calculate_rank() output\n");
    for (int i = sizeof(actualRank) / sizeof(actualRank[0])-1; i >= 0; i--) {
        std::cout << actualRank[i] << ' ' << expectedRank[i] << '\n';
    };

    uint64_t newStakes [4] = {1000000000000000, 5000000000000000, 2000000000000000, 3000000000000000};

    calculate_rank(
        newStakes, stakesSize, cidsSize, linksSize,
        inLinksCount, outLinksCount,
        inLinksOuts, inLinksUsers, outLinksUsers,
        actualRank
    );

    double stakeChangedExpectedRank [6] = {0.1303941838027291, 0.10356462515731377, 0.1640357033015243, 0.17784633379034698, 0.21332336952884043, 0.18703258739196782}

    printf("calculate_rank() output after stake changed\n");
    for (int i = sizeof(actualRank) / sizeof(actualRank[0])-1; i >= 0; i--) {
        std::cout << actualRank[i] << ' ' << stakeChangedExpectedRank[i] << '\n';
    };
}

// To run use `nvcc test_rank.cu -o test && ./test && rm test` command.
int main(void) {
    printf("Start testing !\n");
    
    test_getLinksStartIndex();
    test_getCompressedInLinksCount();
    test_calculateCidTotalOutStake();
    test_find_max_ranks_diff();
    test_getCompressedInLinks();
    test_calculate_rank();

    printf("Finish testing !\n");
}