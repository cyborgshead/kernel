#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include "types.h"

const double DUMP_FACTOR = 0.85;
const double TOLERANCE = 1e-3;

const int CUDA_THREAD_BLOCK_SIZE = 256;

/*****************************************************/
/* KERNEL: RUN SINGLE RANK ITERATION                 */
/*****************************************************/
/* All in links used here are compressed in links    */
/*****************************************************/
__global__
void run_rank_iteration(
    CompressedInLink *inLinks,                            /* all compressed in links */
    double *prevRank, double *rank, uint64_t rankSize,    /* array index - cid index */
    uint64_t *inLinksStartIndex, uint32_t *inLinksCount,  /* array index - cid index */
    double defaultRankWithCorrection // default rank + inner product correction
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < rankSize; i += stride) {

        if(inLinksCount[i] == 0) {
            continue;
        }

        double ksum = 0;
        for (uint64_t j = inLinksStartIndex[i]; j < inLinksStartIndex[i] + inLinksCount[i]; j++) {
           ksum = prevRank[inLinks[j].fromIndex] * inLinks[j].weight + ksum;
           //ksum = __fmaf_rn(prevRank[inLinks[j].fromIndex], inLinks[j].weight, ksum);
        }
        rank[i] = ksum * DUMP_FACTOR + defaultRankWithCorrection;
        //rank[i] = __fmaf_rn(ksum, DUMP_FACTOR, defaultRankWithCorrection);
    }
}


/*****************************************************/
/* KERNEL: DOUBLE ABS FUNCTOR                        */
/*****************************************************/
/* Return absolute value for double                  */
/*****************************************************/
struct absolute_value {
  __device__ double operator()(const double &x) const {
    return x < 0.0 ? -x : x;
  }
};


/*****************************************************/
/* HOST: FINDS MAXIMUM RANKS DIFFERENCE              */
/*****************************************************/
/* Finds maximum rank difference for single element  */
/*                                                   */
/*****************************************************/
double find_max_ranks_diff(double *prevRank, double *newRank, uint64_t rankSize) {

    thrust::device_vector<double> ranksDiff(rankSize);
    thrust::device_ptr<double> newRankBegin(newRank);
    thrust::device_ptr<double> prevRankBegin(prevRank);
    thrust::device_ptr<double> prevRankEnd(prevRank + rankSize);
    thrust::transform(thrust::device,
        prevRankBegin, prevRankEnd, newRankBegin, ranksDiff.begin(), thrust::minus<double>()
    );

    return thrust::transform_reduce(thrust::device,
        ranksDiff.begin(), ranksDiff.end(), absolute_value(), 0.0, thrust::maximum<double>()
    );
}

/*****************************************************/
/* KERNEL: CALCULATE CID TOTAL OUTS STAKE            */
/*****************************************************/
__global__
void calculateCidTotalOutStake(
    uint64_t cidsSize,
    uint64_t *stakes,                                        /*array index - user index*/
    uint64_t *outLinksStartIndex, uint32_t *outLinksCount,   /*array index - cid index*/
    uint64_t *outLinksUsers,                                 /*all out links from all users*/
    /*returns*/ uint64_t *cidsTotalOutStakes                 /*array index - cid index*/
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < cidsSize; i += stride) {
        uint64_t totalOutStake = 0;
        for (uint64_t j = outLinksStartIndex[i]; j < outLinksStartIndex[i] + outLinksCount[i]; j++) {
           totalOutStake += stakes[outLinksUsers[j]];
        }
        cidsTotalOutStakes[i] = totalOutStake;
    }
}

/*********************************************************/
/* KERNEL: CALCULATE COMPRESSED IN LINKS COUNT FOR CIDS  */
/*********************************************************/
__global__
void getCompressedInLinksCount(
    uint64_t cidsSize,
    uint64_t *inLinksStartIndex, uint32_t *inLinksCount,                    /*array index - cid index*/
    uint64_t *inLinksOuts,                                                  /*all incoming links from all users*/
    /*returns*/ uint32_t *compressedInLinksCount                            /*array index - cid index*/
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < cidsSize; i += stride) {

        if(inLinksCount[i] == 0) {
            compressedInLinksCount[i]=0;
            continue;
        }

        uint32_t compressedLinksCount = 0;
        for(uint64_t j = inLinksStartIndex[i]; j < inLinksStartIndex[i]+inLinksCount[i]; j++) {
            if(j == inLinksStartIndex[i] || inLinksOuts[j] != inLinksOuts[j-1]) {
                compressedLinksCount++;
            }
        }
        compressedInLinksCount[i] = compressedLinksCount;
    }
}

/*********************************************************/
/* DEVICE: USER TO DIVIDE TWO uint64                     */
/*********************************************************/
__device__ __forceinline__
double ddiv_rn(uint64_t *a, uint64_t *b) {
    return __ddiv_rn(__ull2double_rn(*a), __ull2double_rn(*b));
}


/*********************************************************/
/* KERNEL: CALCULATE COMPRESSED IN LINKS                 */
/*********************************************************/
__global__
void getCompressedInLinks(
    uint64_t cidsSize,
    uint64_t *inLinksStartIndex, uint32_t *inLinksCount, uint64_t *cidsTotalOutStakes,   /*array index - cid index*/
    uint64_t *inLinksOuts, uint64_t *inLinksUsers,                                       /*all incoming links from all users*/
    uint64_t *stakes,                                                                    /*array index - user index*/
    uint64_t *compressedInLinksStartIndex, uint32_t *compressedInLinksCount,             /*array index - cid index*/
    /*returns*/ CompressedInLink *compressedInLinks                                      /*all incoming compressed links*/
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < cidsSize; i += stride) {

        if(inLinksCount[i] == 0) {
            continue;
        }

        uint32_t compressedLinksIndex = compressedInLinksStartIndex[i];

        if(inLinksCount[i] == 1) {
            uint64_t oppositeCid = inLinksOuts[inLinksStartIndex[i]];
            uint64_t compressedLinkStake = stakes[inLinksUsers[inLinksStartIndex[i]]];
            double weight = ddiv_rn(&compressedLinkStake, &cidsTotalOutStakes[oppositeCid]);
            compressedInLinks[compressedLinksIndex] = CompressedInLink {oppositeCid, weight};
            continue;
        }

        uint64_t compressedLinkStake = 0;
        uint64_t lastLinkIndex = inLinksStartIndex[i] + inLinksCount[i] - 1;
        for(uint64_t j = inLinksStartIndex[i]; j < lastLinkIndex + 1; j++) {

            compressedLinkStake += stakes[inLinksUsers[j]];
            if(j == lastLinkIndex || inLinksOuts[j] != inLinksOuts[j+1]) {
                uint64_t oppositeCid = inLinksOuts[j];
                double weight = ddiv_rn(&compressedLinkStake, &cidsTotalOutStakes[oppositeCid]);
                compressedInLinks[compressedLinksIndex] = CompressedInLink {oppositeCid, weight};
                compressedLinksIndex++;
                compressedLinkStake=0;
            }
        }
    }
}

/************************************************************/
/* HOST: CALCULATE COMPRESSED IN LINKS START INDEXES        */
/************************************************************/
/* SEQUENTIAL LOGIC -> CALCULATE ON CPU                     */
/* RETURNS TOTAL COMPRESSED LINKS SIZE                      */
/************************************************************/
__host__
uint64_t getLinksStartIndex(
    uint64_t cidsSize,
    uint32_t *linksCount,                   /*array index - cid index*/
    /*returns*/ uint64_t *linksStartIndex   /*array index - cid index*/
) {

    uint64_t index = 0;
    for (uint64_t i = 0; i < cidsSize; i++) {
        linksStartIndex[i] = index;
        index += linksCount[i];
    }
    return index;
}

void swap(double* &a, double* &b){
  double *temp = a;
  a = b;
  b = temp;
}

extern "C" {

    void calculate_rank(
        uint64_t *stakes, uint64_t stakesSize,                    /* User stakes and corresponding array size */
        uint64_t cidsSize, uint64_t linksSize,                    /* Cids count */
        uint32_t *inLinksCount, uint32_t *outLinksCount,          /* array index - cid index*/
        uint64_t *inLinksOuts, uint64_t *inLinksUsers,            /*all incoming links from all users*/
        uint64_t *outLinksUsers,                                  /*all outgoing links from all users*/
        double *rank                                              /* array index - cid index*/
    ) {

        // setbuf(stdout, NULL);
        int CUDA_BLOCKS_NUMBER = (cidsSize + CUDA_THREAD_BLOCK_SIZE - 1) / CUDA_THREAD_BLOCK_SIZE;


        // STEP0: Calculate compressed in links start indexes
        /*-------------------------------------------------------------------*/
        // calculated on cpu
        uint64_t *inLinksStartIndex = (uint64_t*) malloc(cidsSize*sizeof(uint64_t));
        uint64_t *outLinksStartIndex = (uint64_t*) malloc(cidsSize*sizeof(uint64_t));
        getLinksStartIndex(cidsSize, inLinksCount, inLinksStartIndex);
        getLinksStartIndex(cidsSize, outLinksCount, outLinksStartIndex);


        // STEP1: Calculate for each cid total stake by out links
        /*-------------------------------------------------------------------*/
        uint64_t *d_outLinksStartIndex;
        uint32_t *d_outLinksCount;
        uint64_t *d_outLinksUsers;
        uint64_t *d_stakes;  // will be used to calculated links weights, should be freed before rank iterations
        uint64_t *d_cidsTotalOutStakes; // will be used to calculated links weights, should be freed before rank iterations

        hipMalloc(&d_outLinksStartIndex, cidsSize*sizeof(uint64_t));
        hipMalloc(&d_outLinksCount,      cidsSize*sizeof(uint32_t));
        hipMalloc(&d_outLinksUsers,     linksSize*sizeof(uint64_t));
        hipMalloc(&d_stakes,           stakesSize*sizeof(uint64_t));
        hipMalloc(&d_cidsTotalOutStakes, cidsSize*sizeof(uint64_t));   //calculated

        hipMemcpy(d_outLinksStartIndex, outLinksStartIndex, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
        hipMemcpy(d_outLinksCount,      outLinksCount,      cidsSize*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(d_outLinksUsers,      outLinksUsers,     linksSize*sizeof(uint64_t), hipMemcpyHostToDevice);
        hipMemcpy(d_stakes,             stakes,           stakesSize*sizeof(uint64_t), hipMemcpyHostToDevice);

        calculateCidTotalOutStake<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_stakes, d_outLinksStartIndex,
            d_outLinksCount, d_outLinksUsers, d_cidsTotalOutStakes
        );

        hipFree(d_outLinksStartIndex);
        hipFree(d_outLinksCount);
        hipFree(d_outLinksUsers);
        /*-------------------------------------------------------------------*/



        // STEP2: Calculate compressed in links count
        /*-------------------------------------------------------------------*/
        uint64_t *d_inLinksStartIndex;
        uint32_t *d_inLinksCount;
        uint64_t *d_inLinksOuts;
        uint32_t *d_compressedInLinksCount;

        // free all before rank iterations
        hipMalloc(&d_inLinksStartIndex,      cidsSize*sizeof(uint64_t));
        hipMalloc(&d_inLinksCount,           cidsSize*sizeof(uint32_t));
        hipMalloc(&d_inLinksOuts,           linksSize*sizeof(uint64_t));
        hipMalloc(&d_compressedInLinksCount, cidsSize*sizeof(uint32_t));   //calculated

        hipMemcpy(d_inLinksStartIndex, inLinksStartIndex, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
        hipMemcpy(d_inLinksCount,      inLinksCount,      cidsSize*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(d_inLinksOuts,       inLinksOuts,      linksSize*sizeof(uint64_t), hipMemcpyHostToDevice);

        getCompressedInLinksCount<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize, d_inLinksStartIndex, d_inLinksCount, d_inLinksOuts, d_compressedInLinksCount
        );
        /*-------------------------------------------------------------------*/



        // STEP3: Calculate compressed in links start indexes
        /*-------------------------------------------------------------------*/
        uint32_t *compressedInLinksCount = (uint32_t*) malloc(cidsSize*sizeof(uint32_t));
        uint64_t *compressedInLinksStartIndex = (uint64_t*) malloc(cidsSize*sizeof(uint64_t));
        hipMemcpy(compressedInLinksCount, d_compressedInLinksCount, cidsSize * sizeof(uint32_t), hipMemcpyDeviceToHost);

        // calculated on cpu
        uint64_t compressedInLinksSize = getLinksStartIndex(
            cidsSize, compressedInLinksCount, compressedInLinksStartIndex
        );

        uint64_t *d_compressedInLinksStartIndex;
        hipMalloc(&d_compressedInLinksStartIndex, cidsSize*sizeof(uint64_t));
        hipMemcpy(d_compressedInLinksStartIndex, compressedInLinksStartIndex, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
        free(compressedInLinksStartIndex);
        /*-------------------------------------------------------------------*/



        // STEP4: Calculate compressed in links
        /*-------------------------------------------------------------------*/
        uint64_t *d_inLinksUsers;
        CompressedInLink *d_compressedInLinks; //calculated

        hipMalloc(&d_inLinksUsers,                   linksSize*sizeof(uint64_t));
        hipMalloc(&d_compressedInLinks,  compressedInLinksSize*sizeof(CompressedInLink));
        hipMemcpy(d_inLinksUsers, inLinksUsers,      linksSize*sizeof(uint64_t), hipMemcpyHostToDevice);

        getCompressedInLinks<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
            cidsSize,
            d_inLinksStartIndex, d_inLinksCount, d_cidsTotalOutStakes,
            d_inLinksOuts, d_inLinksUsers, d_stakes,
            d_compressedInLinksStartIndex, d_compressedInLinksCount,
            d_compressedInLinks
        );

        hipFree(d_inLinksUsers);
        hipFree(d_inLinksStartIndex);
        hipFree(d_inLinksCount);
        hipFree(d_inLinksOuts);
        hipFree(d_stakes);
        hipFree(d_cidsTotalOutStakes);
        /*-------------------------------------------------------------------*/



        // STEP5: Calculate dangling nodes rank, and default rank
        /*-------------------------------------------------------------------*/
        double defaultRank = (1.0 - DUMP_FACTOR) / cidsSize;
        uint64_t danglingNodesSize = 0;
        for(uint64_t i=0; i< cidsSize; i++){
            rank[i] = defaultRank;
            if(inLinksCount[i] == 0) {
                danglingNodesSize++;
            }
        }

        double innerProductOverSize = defaultRank * ((double) danglingNodesSize / (double)cidsSize);
        double defaultRankWithCorrection = (DUMP_FACTOR * innerProductOverSize) + defaultRank; //fma point
        /*-------------------------------------------------------------------*/




        // STEP6: Calculate rank
        /*-------------------------------------------------------------------*/
        double *d_rank, *d_prevRank;

        hipMalloc(&d_rank, cidsSize*sizeof(double));
        hipMalloc(&d_prevRank, cidsSize*sizeof(double));

        hipMemcpy(d_rank,     rank, cidsSize*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_prevRank, rank, cidsSize*sizeof(double), hipMemcpyHostToDevice);

        int steps = 0;
        double change = TOLERANCE + 1.0;
        while(change > TOLERANCE) {
            swap(d_rank, d_prevRank);
            steps++;
        	run_rank_iteration<<<CUDA_BLOCKS_NUMBER,CUDA_THREAD_BLOCK_SIZE>>>(
                d_compressedInLinks,
                d_prevRank, d_rank, cidsSize,
                d_compressedInLinksStartIndex, d_compressedInLinksCount,
                defaultRankWithCorrection
        	);
        	change = find_max_ranks_diff(d_prevRank, d_rank, cidsSize);
        	hipDeviceSynchronize();
        }

        hipMemcpy(rank, d_rank, cidsSize * sizeof(double), hipMemcpyDeviceToHost);
        /*-------------------------------------------------------------------*/


        hipFree(d_rank);
        hipFree(d_prevRank);
        hipFree(d_compressedInLinksStartIndex);
        hipFree(d_compressedInLinksCount);
        hipFree(d_compressedInLinks);
    }
};
